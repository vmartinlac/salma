#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/imgcodecs.hpp>
#include "StereoMatcherImpl.h"

__constant__ int cte_margin = 0;
__constant__ int cte_num_disparities = 0;
__constant__ int cte_width = 0;
__constant__ int cte_height = 0;
__constant__ int cte_neighbor_dx[4] = { 1, 0, -1, 0 };
__constant__ int cte_neighbor_dy[4] = { 0, 1, 0, -1 };

__device__ size_t occlusion_message_index(int xfrom, int yfrom, int neighbor, int msg)
{
    return 4*2*cte_width*yfrom + 4*2*xfrom + 2*neighbor + msg;
}

__device__ size_t disparity_message_index(int xfrom, int yfrom, int neighbor, int msg)
{
    return 4*cte_num_disparities*cte_width*yfrom + 4*cte_num_disparities*xfrom + cte_num_disparities*neighbor + msg;
}

__device__ bool isInROI(int x, int y)
{
    return ( cte_margin <= x && x < cte_width - cte_margin && cte_margin <= y && y < cte_height - cte_margin );
}

__global__ void kernel_initialize_disparity_and_occlusion(
    cv::cuda::PtrStep<short1> disparity_left,
    cv::cuda::PtrStep<short1> disparity_right,
    cv::cuda::PtrStep<uchar1> occlusion_left,
    cv::cuda::PtrStep<uchar1> occlusion_right)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if( x < cte_width && y < cte_height )
    {
        disparity_left(y,x) = make_short1(0);
        disparity_right(y,x) = make_short1(0);
        occlusion_left(y,x) = make_uchar1(0);
        occlusion_right(y,x) = make_uchar1(0);
    }
}

__global__ void kernel_clear_warp_and_messages(
    cv::cuda::PtrStep<uchar1> warp_left,
    cv::cuda::PtrStep<uchar1> warp_right,
    float* msg_occlusion_left,
    float* msg_occlusion_right,
    float* msg_disparity_left,
    float* msg_disparity_right)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if( x < cte_width && y < cte_height)
    {
        warp_left(y,x) = make_uchar1(1);
        warp_right(y,x) = make_uchar1(1);

        for(int i=0; i<4; i++)
        {
            for(int j=0; j<2; j++)
            {
                msg_occlusion_left[occlusion_message_index(x, y, i, j)] = 0.0;
                msg_occlusion_right[occlusion_message_index(x, y, i, j)] = 0.0;
            }

            for(int j=0; j<cte_num_disparities; j++)
            {
                msg_disparity_left[disparity_message_index(x, y, i, j)] = 0.0;
                msg_disparity_right[disparity_message_index(x, y, i, j)] = 0.0;
            }
        }
    }
}

__global__ void kernel_compute_warp(
    cv::cuda::PtrStep<short1> disparity_left,
    cv::cuda::PtrStep<short1> disparity_right,
    cv::cuda::PtrStep<uchar1> occlusion_left,
    cv::cuda::PtrStep<uchar1> occlusion_right,
    cv::cuda::PtrStep<uchar1> warp_left,
    cv::cuda::PtrStep<uchar1> warp_right)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if( isInROI(x,y) )
    {
        // compute left warp.

        if( occlusion_right(y,x).x == 0 )
        {
            const int left_x = x + disparity_right(y,x).x;

            if( cte_margin <= left_x && left_x < cte_width - cte_margin )
            {
                warp_left(y,left_x) = make_uchar1(0);
            }
        }

        // compute right warp.

        if( occlusion_left(y,x).x == 0 )
        {
            const int right_x = x + disparity_left(y,x).x;

            if( cte_margin <= right_x && right_x < cte_width - cte_margin )
            {
                warp_right(y,right_x) = make_uchar1(0);
            }
        }
    }
}

__global__ void kernel_occlusion_iteration(
    cv::cuda::PtrStep<uchar3> image_left,
    cv::cuda::PtrStep<uchar3> image_right,
    cv::cuda::PtrStep<uchar1> warp_left,
    cv::cuda::PtrStep<uchar1> warp_right,
    cv::cuda::PtrStep<short1> disparity_left,
    cv::cuda::PtrStep<short1> disparity_right,
    float* occlusion_messages_left_pre,
    float* occlusion_messages_right_pre,
    float* occlusion_messages_left_post,
    float* occlusion_messages_right_post)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if( isInROI(x,y) )
    {
        for(int i=0; i<4; i++)
        {
            const int xo = x + cte_neighbor_dx[i];
            const int yo = y + cte_neighbor_dy[i];

            if( isInROI(xo, yo) )
            {
                for(int fo=0; fo<2; fo++)
                {
                    float best_value_left = 0.0f;
                    float best_value_right = 0.0f;

                    for(int f=0; f<2; f++)
                    {
                        float value_left = 0.0f;
                        float value_right = 0.0f;

                        // TODO

                        if(f == 0 || value_left < best_value_left)
                        {
                            best_value_left = value_left;
                        }

                        if(f == 0 || value_right < best_value_right)
                        {
                            best_value_right = value_right;
                        }
                    }

                    occlusion_messages_left_post[occlusion_message_index(x, y, i, fo)] = best_value_left;
                    occlusion_messages_right_post[occlusion_message_index(x, y, i, fo)] = best_value_right;
                }
            }
        }
    }
}

__global__ void kernel_occlusion_update()
{
}

__global__ void kernel_disparity_iteration()
{
}

__global__ void kernel_disparity_update()
{
}

StereoMatcherImpl::StereoMatcherImpl()
{
    mNumDisparities = 16;
    mMargin = 1;
}

void StereoMatcherImpl::compute(cv::InputArray left, cv::InputArray right, cv::OutputArray disparity)
{
    const int num_iterations_for_occlusion_belief_propagation = 8;
    const int num_iterations_for_disparity_belief_propagation = 8;
    const int num_iterations_for_fixed_point = 4;

    cv::cuda::GpuMat d_image_left = left.getGpuMat();
    cv::cuda::GpuMat d_image_right = right.getGpuMat();

    if( d_image_left.size() != d_image_right.size() ) throw std::runtime_error("left and right images should be the same size!");

    const cv::Size2i size = d_image_left.size();

    if( size.width <= 2*mMargin || size.height <= 2*mMargin ) throw std::runtime_error("image is too small.");

    cv::cuda::GpuMat d_occlusion_left;
    d_occlusion_left.create( size, CV_8UC1 );

    cv::cuda::GpuMat d_occlusion_right;
    d_occlusion_right.create( size, CV_8UC1 );

    cv::cuda::GpuMat d_disparity_left;
    d_disparity_left.create( size, CV_16SC1 );

    cv::cuda::GpuMat d_disparity_right;
    d_disparity_right.create( size, CV_16SC1 );

    cv::cuda::GpuMat d_warp_left;
    d_warp_left.create( size, CV_8UC1 );

    cv::cuda::GpuMat d_warp_right;
    d_warp_right.create( size, CV_8UC1 );

    float* d_occlusion_messages_left[2];
    float* d_occlusion_messages_right[2];
    float* d_disparity_messages_left[2];
    float* d_disparity_messages_right[2];
    cudaSafeCall( hipMalloc(&d_occlusion_messages_left[0], size.width*size.height*2*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_occlusion_messages_left[1], size.width*size.height*2*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_occlusion_messages_right[0], size.width*size.height*2*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_occlusion_messages_right[1], size.width*size.height*2*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_disparity_messages_left[0], size.width*size.height*mNumDisparities*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_disparity_messages_left[1], size.width*size.height*mNumDisparities*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_disparity_messages_right[0], size.width*size.height*mNumDisparities*4*sizeof(float)) );
    cudaSafeCall( hipMalloc(&d_disparity_messages_right[1], size.width*size.height*mNumDisparities*4*sizeof(float)) );

    const dim3 block_dim( 16, 16 );
    const dim3 grid_dim( (size.width + block_dim.x - 1) / block_dim.x, (size.height + block_dim.y - 1) / block_dim.y );

    cv::cuda::Stream stream;
    const hipStream_t stream_id = cv::cuda::StreamAccessor::getStream(stream);

    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cte_margin), &mMargin, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cte_num_disparities), &mNumDisparities, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cte_width), &size.width, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cte_height), &size.height, sizeof(int)) );

    kernel_initialize_disparity_and_occlusion<<<grid_dim, block_dim, 0, stream_id>>>(
        d_disparity_left,
        d_disparity_right,
        d_occlusion_left,
        d_occlusion_right);

    for(int i=0; i<num_iterations_for_fixed_point; i++)
    {
        kernel_clear_warp_and_messages<<<grid_dim, block_dim, 0, stream_id>>>(
            d_warp_left,
            d_warp_right,
            d_occlusion_messages_left[0],
            d_occlusion_messages_right[0],
            d_disparity_messages_left[0],
            d_disparity_messages_right[0]);

        kernel_compute_warp<<<grid_dim, block_dim, 0, stream_id>>>(
            d_disparity_left,
            d_disparity_right,
            d_occlusion_left,
            d_occlusion_right,
            d_warp_left,
            d_warp_right);

        for(int j=0; j<num_iterations_for_occlusion_belief_propagation; j++)
        {
            kernel_occlusion_iteration<<<grid_dim, block_dim, 0, stream_id>>>(
                d_image_left,
                d_image_right,
                d_warp_left,
                d_warp_right,
                d_disparity_left,
                d_disparity_right,
                d_occlusion_messages_left[0],
                d_occlusion_messages_right[0],
                d_occlusion_messages_left[1],
                d_occlusion_messages_right[1]);

            std::swap( d_occlusion_messages_left[0], d_occlusion_messages_left[1] );
            std::swap( d_occlusion_messages_right[0], d_occlusion_messages_right[1] );
        }

        kernel_occlusion_update<<<grid_dim, block_dim, 0, stream_id>>>();

        for(int j=0; j<num_iterations_for_disparity_belief_propagation; j++)
        {
            kernel_disparity_iteration<<<grid_dim, block_dim, 0, stream_id>>>();
                /*
                d_image_left,
                d_image_right,
                d_warp_left,
                d_warp_right,
                d_disparity_left,
                d_disparity_right,
                d_disparity_messages_left[0],
                d_disparity_messages_right[0],
                d_disparity_messages_left[1],
                d_disparity_messages_right[1]);
                */

            std::swap( d_disparity_messages_left[0], d_disparity_messages_left[1] );
            std::swap( d_disparity_messages_right[0], d_disparity_messages_right[1] );
        }

        kernel_disparity_update<<<grid_dim, block_dim, 0, stream_id>>>();
    }

    stream.waitForCompletion();

    disparity.getGpuMatRef() = d_disparity_left;

    cudaSafeCall( hipFree(d_occlusion_messages_left[0]) );
    cudaSafeCall( hipFree(d_occlusion_messages_left[1]) );
    cudaSafeCall( hipFree(d_occlusion_messages_right[0]) );
    cudaSafeCall( hipFree(d_occlusion_messages_right[1]) );
    cudaSafeCall( hipFree(d_disparity_messages_left[0]) );
    cudaSafeCall( hipFree(d_disparity_messages_left[1]) );
    cudaSafeCall( hipFree(d_disparity_messages_right[0]) );
    cudaSafeCall( hipFree(d_disparity_messages_right[1]) );
}

StereoMatcher* StereoMatcher::create()
{
    return new StereoMatcherImpl();
}

int StereoMatcherImpl::getMinDisparity() const
{
    return 0;
}


void StereoMatcherImpl::setMinDisparity(int minDisparity)
{
    // ignored
}


int StereoMatcherImpl::getNumDisparities() const
{
    return mNumDisparities;
}


void StereoMatcherImpl::setNumDisparities(int numDisparities)
{
    mNumDisparities = numDisparities;
}


int StereoMatcherImpl::getBlockSize() const
{
    return 0;
}


void StereoMatcherImpl::setBlockSize(int blockSize)
{
    // ignored
}


int StereoMatcherImpl::getSpeckleWindowSize() const
{
    return 0;
}


void StereoMatcherImpl::setSpeckleWindowSize(int speckleWindowSize)
{
    // ignored
}


int StereoMatcherImpl::getSpeckleRange() const
{
    return 0;
}


void StereoMatcherImpl::setSpeckleRange(int speckleRange)
{
    // ignored
}


int StereoMatcherImpl::getDisp12MaxDiff() const
{
    return 0;
}


void StereoMatcherImpl::setDisp12MaxDiff(int disp12MaxDiff)
{
    // ignored
}


