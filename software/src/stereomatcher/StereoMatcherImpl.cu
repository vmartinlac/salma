#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/imgcodecs.hpp>
#include "StereoMatcherImpl.h"

__global__ void hello(cv::cuda::PtrStepSz<uchar3> image)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < image.cols && y < image.rows)
    {
        x+y;
    }
}


StereoMatcherImpl::StereoMatcherImpl()
{
    mNumDisparities = 16;
}

void StereoMatcherImpl::compute(cv::InputArray left, cv::InputArray right, cv::OutputArray disparity)
{
    cv::cuda::GpuMat d_occlusion_left;
    cv::cuda::GpuMat d_occlusion_right;

    cv::cuda::GpuMat d_disparity_left;
    cv::cuda::GpuMat d_disparity_right;

    cv::cuda::Stream stream_left;
    cv::cuda::Stream stream_right;

    cv::cuda::GpuMat d_image_left = left.getGpuMat();
    cv::cuda::GpuMat d_image_right = right.getGpuMat();

    if( d_image_left.size() != d_image_right.size() ) throw std::runtime_error("left and right images should be the same size!");

    const cv::Size size = d_image_left.size();

    d_occlusion_left.create( size, CV_8UC1 );
    d_occlusion_right.create( size, CV_8UC1 );

    d_disparity_left.create( size, CV_16SC1 );
    d_disparity_right.create( size, CV_16SC1 );

    const dim3 block_dim( 16, 16 );
    const dim3 grid_dim( (size.width + block_dim.x - 1) / block_dim.x, (size.height + block_dim.y - 1) / block_dim.y );

    const hipStream_t cuda_stream_left = cv::cuda::StreamAccessor::getStream(stream_left);
    const hipStream_t cuda_stream_right = cv::cuda::StreamAccessor::getStream(stream_right);

    const int num_iterations_for_occlusion_belief_propagation = 5;
    const int num_iterations_for_disparity_belief_propagation = 5;
    const int num_iterations_for_fixed_point = 3;

    std::vector<cv::cuda::Event> occlusion_events_left(num_iterations_for_fixed_point);
    std::vector<cv::cuda::Event> occlusion_events_right(num_iterations_for_fixed_point);
    std::vector<cv::cuda::Event> disparity_events_left(num_iterations_for_fixed_point);
    std::vector<cv::cuda::Event> disparity_events_right(num_iterations_for_fixed_point);

    for(int i=0; i<num_iterations_for_fixed_point; i++)
    {
        for(int j=0; j<num_iterations_for_occlusion_belief_propagation; j++)
        {
            hello<<<grid_dim, block_dim, 0, cuda_stream_left>>>(d_image_left);
            hello<<<grid_dim, block_dim, 0, cuda_stream_right>>>(d_image_right);
        }

        occlusion_events_left[i].record(stream_left);
        occlusion_events_right[i].record(stream_right);

        stream_left.waitEvent( occlusion_events_right[i] );
        stream_right.waitEvent( occlusion_events_left[i] );

        for(int j=0; j<num_iterations_for_disparity_belief_propagation; j++)
        {
            hello<<<grid_dim, block_dim, 0, cuda_stream_left>>>(d_image_left);
            hello<<<grid_dim, block_dim, 0, cuda_stream_right>>>(d_image_right);
        }

        occlusion_events_left[i].record(stream_left);
        occlusion_events_right[i].record(stream_right);

        stream_left.waitEvent( disparity_events_right[i] );
        stream_right.waitEvent( disparity_events_left[i] );
    }

    stream_left.waitForCompletion();
    stream_right.waitForCompletion();

    // TODO retrieve resulting disparity.
}

StereoMatcher* StereoMatcher::create()
{
    return new StereoMatcherImpl();
}

int StereoMatcherImpl::getMinDisparity() const
{
    return 0;
}


void StereoMatcherImpl::setMinDisparity(int minDisparity)
{
    // ignored
}


int StereoMatcherImpl::getNumDisparities() const
{
    return mNumDisparities;
}


void StereoMatcherImpl::setNumDisparities(int numDisparities)
{
    mNumDisparities = numDisparities;
}


int StereoMatcherImpl::getBlockSize() const
{
    return 0;
}


void StereoMatcherImpl::setBlockSize(int blockSize)
{
    // ignored
}


int StereoMatcherImpl::getSpeckleWindowSize() const
{
    return 0;
}


void StereoMatcherImpl::setSpeckleWindowSize(int speckleWindowSize)
{
    // ignored
}


int StereoMatcherImpl::getSpeckleRange() const
{
    return 0;
}


void StereoMatcherImpl::setSpeckleRange(int speckleRange)
{
    // ignored
}


int StereoMatcherImpl::getDisp12MaxDiff() const
{
    return 0;
}


void StereoMatcherImpl::setDisp12MaxDiff(int disp12MaxDiff)
{
    // ignored
}


