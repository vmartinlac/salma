#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgcodecs.hpp>
#include "StereoMatcherImpl.h"

__global__ void hello(
    cv::cuda::PtrStepSz<uchar3> left,
    cv::cuda::PtrStepSz<uchar3> right,
    cv::cuda::PtrStepSz<uchar3> image)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < image.cols && y < image.rows)
    {
        uchar3 l = left(y,x);
        uchar3 r = right(y,x);
    }
}


StereoMatcherImpl::StereoMatcherImpl()
{
    mNumDisparities = 16;
}

void StereoMatcherImpl::compute(cv::InputArray left, cv::InputArray right, cv::OutputArray disparity)
{
    cv::cuda::GpuMat d_left = left.getGpuMat();
    cv::cuda::GpuMat d_right = right.getGpuMat();
    cv::cuda::GpuMat& d_disp = disparity.getGpuMatRef();

    const int width = d_left.cols;
    const int height = d_left.rows;

    if( width != d_right.cols || height != d_right.rows ) throw std::runtime_error("left and right images should be the same size!");

    mBlockWidth = 16;
    mBlockHeight = 16;
    mGridWidth = (width + mBlockWidth - 1) / mBlockWidth;
    mGridHeight = (height + mBlockHeight - 1) / mBlockHeight;

    d_disp.create( width, height, CV_8UC3 );

    hello<<< dim3(mGridWidth,mGridHeight), dim3(mBlockWidth,mBlockHeight) >>>(d_left, d_right, d_disp);

    hipDeviceSynchronize();
}

StereoMatcher* StereoMatcher::create()
{
    return new StereoMatcherImpl();
}

int StereoMatcherImpl::getMinDisparity() const
{
    return 0;
}


void StereoMatcherImpl::setMinDisparity(int minDisparity)
{
    // ignored
}


int StereoMatcherImpl::getNumDisparities() const
{
    return mNumDisparities;
}


void StereoMatcherImpl::setNumDisparities(int numDisparities)
{
    mNumDisparities = numDisparities;
}


int StereoMatcherImpl::getBlockSize() const
{
    return 0;
}


void StereoMatcherImpl::setBlockSize(int blockSize)
{
    // ignored
}


int StereoMatcherImpl::getSpeckleWindowSize() const
{
    return 0;
}


void StereoMatcherImpl::setSpeckleWindowSize(int speckleWindowSize)
{
    // ignored
}


int StereoMatcherImpl::getSpeckleRange() const
{
    return 0;
}


void StereoMatcherImpl::setSpeckleRange(int speckleRange)
{
    // ignored
}


int StereoMatcherImpl::getDisp12MaxDiff() const
{
    return 0;
}


void StereoMatcherImpl::setDisp12MaxDiff(int disp12MaxDiff)
{
    // ignored
}


